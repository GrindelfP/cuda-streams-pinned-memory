#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>

/**
* Kernel function for the program.
*/
__global__
void
kernel(
    float* dA,
    float* dB,
    float* dC,
    int size
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j;
    float ab, sum = 0.f;

    if (i < size) {
        ab = dA[i] * dB[i];

        for (j = 0; j < 1000; ++j) sum += sinf(j + ab);

        dC[i] = sum;
    }
}

/**
* CPU function for the program.
*/
void
cpu_compute(
    float* hA,
    float* hB,
    float* hC,
    int size
) {
    for (int i = 0; i < size; ++i) {
        float ab = hA[i] * hB[i];
        float sum = 0.f;

        for (int j = 0; j < 1000; ++j)  sum += sinf(j + ab);

        hC[i] = sum;
    }
}

void
testGPU(
    const int nStreams,
    float* hA,
    float *hB,
    float *hC_GPU, 
    std::chrono::duration<float, std::milli> cpuDuration
) {
    // ===================
    // DATA INITIALIZATION
    // ===================
    const int nThreads = 512, totalSize = 512 * 50000, size = totalSize / nStreams;
    const size_t memSize = size * sizeof(float), arraySize = totalSize * sizeof(float);

    float* dA, * dB, * dC;

    hipMalloc((void**)&dA, arraySize);
    hipMalloc((void**)&dB, arraySize);
    hipMalloc((void**)&dC, arraySize);

    // ================================== 
    // GPU computation and data maagement
    // ================================== 
    hipStream_t *streams = new hipStream_t[nStreams];

    for (int i = 0; i < nStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }

    int nBlocks = (size + nThreads - 1) / nThreads;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < nStreams; ++i) {
        hipMemcpyAsync(dA + i * size, hA + i * size, memSize, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(dB + i * size, hB + i * size, memSize, hipMemcpyHostToDevice, streams[i]);
    }

    for (int i = 0; i < nStreams; ++i) {
        kernel << <nBlocks, nThreads, 0, streams[i] >> > (dA + i * size, dB + i * size, dC + i * size, size);
    }

    for (int i = 0; i < nStreams; ++i) {
        hipMemcpyAsync(hC_GPU + i * size, dC + i * size, memSize, hipMemcpyDeviceToHost, streams[i]);
    }

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float gpuTime;
    hipEventElapsedTime(&gpuTime, start, stop);
    std::cout << "GPU operation done with number of streams: " << nStreams << ".\n";

    std::cout << "GPU calculation time: " << gpuTime << " ms\n";
    std::cout << "Rate: " << cpuDuration.count() / gpuTime << "x\n";

    // ========
    // CLEAN UP
    // ========
    for (int i = 0; i < nStreams; ++i) {
        hipStreamDestroy(streams[i]);
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    delete[] streams;
}

/**
* Main function for the program.
*/
int
main() {
    // ===================
    // DATA INITIALIZATION
    // ===================
    const int streamsNumbers[] = { 1, 2, 4 };
    const int nThreads = 512, totalSize = 512 * 50000;
    const size_t arraySize = totalSize * sizeof(float);

    float* hA, * hB, * hC, * hC_GPU;

    hipHostMalloc((void**)&hA, arraySize);
    hipHostMalloc((void**)&hB, arraySize);
    hipHostMalloc((void**)&hC, arraySize);
    hipHostMalloc((void**)&hC_GPU, arraySize);

    for (int i = 0; i < totalSize; ++i) {
        hA[i] = sinf(i);
        hB[i] = cosf(2.0f * i - 5.0f);
        hC[i] = 0.0f;
        hC_GPU[i] = 0.0f;
    }

    // ===============
    // CPU COMPUTATION
    // ===============
    auto cpuStart = std::chrono::high_resolution_clock::now();
    cpu_compute(hA, hB, hC, totalSize);
    auto cpuStop = std::chrono::high_resolution_clock::now();

    std::chrono::duration<float, std::milli> cpuDuration = cpuStop - cpuStart;
    std::cout << "CPU operation done!\n";
    std::cout << "CPU calculation time: " << cpuDuration.count() << " ms\n";
    std::cout << "Number of streams: 1\n\n";

    // ===============
    // GPU COMPUTATION
    // ===============
    for (int i = 0; i < 3; ++i) testGPU(streamsNumbers[i], hA, hB, hC_GPU, cpuDuration);

    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipHostFree(hC_GPU);

    return 0;
}
